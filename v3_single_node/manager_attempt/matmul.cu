#include <iostream>
#include <hip/hip_runtime.h>

// Define the size of the matrices.  Adjust as needed.
#define N 1024

// CUDA kernel for matrix multiplication
__global__ void matrixMulKernel(float *A, float *B, float *C, int n) {
    // Calculate the row and column index for this thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread is within the bounds of the matrix
    if (row < n && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; ++k) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    // Host memory allocation
    float *h_A = new float[N * N];
    float *h_B = new float[N * N];
    float *h_C = new float[N * N];

    // Initialize matrices A and B with some values
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = (float)i;
        h_B[i] = (float)(N * N - i);
    }

    // Device memory allocation
    float *d_A;
    float *d_B;
    float *d_C;
    hipMalloc((void **)&d_A, N * N * sizeof(float));
    hipMalloc((void **)&d_B, N * N * sizeof(float));
    hipMalloc((void **)&d_C, N * N * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Define the grid and block dimensions
    dim3 blockDim(16, 16); // Adjust block size as needed for optimal performance
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

    // Launch the kernel
    matrixMulKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);

    // Copy the result from device to host
    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Verify the result (optional)
    // You can compare h_C with the expected result here.

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    std::cout << "Matrix multiplication completed successfully." << std::endl;

    return 0;
}
