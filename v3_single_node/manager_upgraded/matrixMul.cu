#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel for matrix multiplication
__global__ void matrixMulKernel(float *A, float *B, float *C, int n) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < n && col < n) {
    float sum = 0.0f;
    for (int k = 0; k < n; ++k) {
      sum += A[row * n + k] * B[k * n + col];
    }
    C[row * n + col] = sum;
  }
}

int main() {
  int n = 256; // Matrix size
  size_t matrixSize = n * n * sizeof(float);

  // Allocate host memory
  float *h_A = new float[n * n];
  float *h_B = new float[n * n];
  float *h_C = new float[n * n];

  // Initialize matrices A and B (example initialization)
  for (int i = 0; i < n * n; ++i) {
    h_A[i] = 1.0f;
    h_B[i] = 2.0f;
  }

  // Allocate device memory
  float *d_A, *d_B, *d_C;
  hipMalloc((void **)&d_A, matrixSize);
  hipMalloc((void **)&d_B, matrixSize);
  hipMalloc((void **)&d_C, matrixSize);

  // Copy data from host to device
  hipMemcpy(d_A, h_A, matrixSize, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, matrixSize, hipMemcpyHostToDevice);

  // Define grid and block dimensions
  dim3 blockDim(16, 16);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);

  // Launch the kernel
  matrixMulKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, n);

  // Copy data from device to host
  hipMemcpy(h_C, d_C, matrixSize, hipMemcpyDeviceToHost);

  // Verify the result (optional)
  std::cout << "C[0][0] = " << h_C[0] << std::endl;

  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  // Free host memory
  delete[] h_A;
  delete[] h_B;
  delete[] h_C;

  return 0;
}
